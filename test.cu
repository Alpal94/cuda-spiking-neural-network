#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int** arr)
{
	for (int i=0; i<3; i++)
		printf("%d\n", arr[i][0]);
}

int main()
{
	int arr[][3] = {{1},{2},{3}}; // 3 arrays, 1 element each
	char x = 0b100000100;
	printf("Bin: %d\n", x);

	int **d_arr;

	hipMalloc((void***)(&d_arr), sizeof(int*)*3); // allocate for 3 int pointers

	for (int i=0; i<3; i++)
	{
		int* temp;
		hipMalloc( (void**)  &(temp), sizeof(int) * 1 ); // allocate for 1 int in each int pointer
		hipMemcpy(temp, arr[i], sizeof(int) * 1, hipMemcpyHostToDevice); // copy data
		hipMemcpy(d_arr+i, &temp, sizeof(int*), hipMemcpyHostToDevice);
	}

	kernel<<<1,1>>>(d_arr);
	hipDeviceSynchronize();
	hipDeviceReset();
}
